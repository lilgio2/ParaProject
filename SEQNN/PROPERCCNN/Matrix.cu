#include "Matrix.cuh"
#include <random>

Matrix::Matrix()
{

}


Matrix::Matrix(int _numRows, int _numCols, bool isRandom)
{
	numRows = _numRows;
	numCols = _numCols;

	double r = 0.00;

	vector<double> colValues;

	for (int i = 0; i < numRows; i++)
	{

		for (int j = 0; j < numCols; j++)
		{
			if (isRandom)
			{
				r = generateRandomNumber();
			}

			colValues.push_back(r);
		}

		values.push_back(colValues);
		colValues.clear();
	}
}

double Matrix::generateRandomNumber()
{
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> dis(0, 1);

	return dis(gen);
}

void Matrix::printToConsole()
{
	for (int i = 0; i < numRows; i++)
	{
		for (int j = 0; j < numCols; j++)
		{
			cout << this->values.at(i).at(j) << "\t\t";
		}
		cout << endl;
	}
}

Matrix Matrix::transpose()
{
	Matrix m = Matrix(numCols, numRows, false);

	for (int i = 0; i < numRows; ++i)
	{
		for (int j = 0; j < numCols; ++j)
		{
			m.setValue(j, i, getValue(i, j));
		}
	}
	return m;
}