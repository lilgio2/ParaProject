#include "MultiplyMatrix.cuh"
#include <math.h>

utils::MultiplyMatrix::MultiplyMatrix(Matrix _a, Matrix _b)
{
	a = _a;
	b = _b;

	if (a.getNumCols() != b.getNumRows()) {
		cerr << "A_rows: " << a.getNumRows() << " != B_cols: " << b.getNumCols() << endl;
		assert(false);
	}

	c = Matrix(a.getNumRows(), b.getNumCols(), false);
}

Matrix utils::MultiplyMatrix::execute()
{
	
	for (int i = 0; i < a.getNumRows(); ++i)
	{
		for (int j = 0; j < b.getNumCols(); ++j)
		{
			for (int k = 0; k < b.getNumRows(); ++k)
			{
				double p = a.getValue(i, k) * b.getValue(k, j);
				double newVal = c.getValue(i, j) + p;
				c.setValue(i, j, newVal);
			}
		}
	}
	
	return c;
}



