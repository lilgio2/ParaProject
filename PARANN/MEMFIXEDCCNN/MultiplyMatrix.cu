#include "hip/hip_runtime.h"
#include "MultiplyMatrix.cuh"
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void matrixmult(double* a, double* b, double* c, int thicc, int nice)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	for (int x = 0; x < nice; ++x)
	{
		c[i * thicc + j] += a[i * nice + x] * b[x * thicc + j];
	}


	//rcsi
	//icsc
}

utils::MultiplyMatrix::MultiplyMatrix(Matrix _a, Matrix _b)
{
	a = _a;
	b = _b;

	if (a.getNumCols() != b.getNumRows()) {
		cerr << "A_rows: " << a.getNumRows() << " != B_cols: " << b.getNumCols() << endl;
		assert(false);
	}

	c = Matrix(a.getNumRows(), b.getNumCols(), false);
}

Matrix utils::MultiplyMatrix::execute()
{
	// + 2 for a memory buffer
	int AArraySize = a.getNumRows() * a.getNumCols() + 2;
	int BArraySize = b.getNumRows() * b.getNumCols() + 2;
	int CArraySize = c.getNumRows() * c.getNumCols() + 2;


	double* A = new double[AArraySize];
	double* B = new double[BArraySize];
	double* C = new double[CArraySize];

	int indexcheck = 0;
	for (int i = 0; i < a.getNumRows(); ++i)
	{
		for (int j = 0; j < a.getNumCols(); ++j)
		{
			A[i * a.getNumCols() + j] = a.getValue(i, j);
		}
	}

	for (int i = 0; i < b.getNumRows(); ++i)
	{
		for (int j = 0; j < b.getNumCols(); ++j)
		{
			B[i * b.getNumCols() + j] = b.getValue(i, j);
		}
	}
	
	for (int i = 0; i < c.getNumRows(); ++i)
	{
		for (int j = 0; j < c.getNumCols(); ++j)
		{
			C[i * c.getNumCols() + j] = c.getValue(i, j);
		}
	}
	
	//BEGIN BLOCK B
	
	double* DA;
	hipMalloc((void **) &DA, sizeof(double) * AArraySize);
	
	double* DB;
	hipMalloc((void**) &DB, sizeof(double) * BArraySize);
	
	double* DC;
	hipMalloc((void**) &DC, sizeof(double) * CArraySize);

	hipMemcpy(DA, A, sizeof(double) * AArraySize, hipMemcpyHostToDevice);
	hipMemcpy(DB, B, sizeof(double) * BArraySize, hipMemcpyHostToDevice);
	hipMemcpy(DC, C, sizeof(double) * CArraySize, hipMemcpyHostToDevice);
	
	dim3 dimGrid(1, 1);
	dim3 dimBlock(a.getNumRows(), b.getNumCols());
	int thicc = b.getNumCols();
	int nice = b.getNumRows();
	
	matrixmult<<<dimGrid, dimBlock >>>(DA, DB, DC, thicc, nice);

	hipMemcpy(C, DC, sizeof(double) * CArraySize, hipMemcpyDeviceToHost);

	for (int i = 0; i < c.getNumRows(); ++i)
	{
		for (int j = 0; j < c.getNumCols(); ++j)
		{
			c.setValue(i,j, C[i * c.getNumCols() + j]);
		}
	}	
	
	hipFree(DA);
	hipFree(DB);
	hipFree(DC);

	delete[] A;
	delete[] B;
	delete[] C;

	return c;
}



