#include "hip/hip_runtime.h"
#include "NeuralNetwork.cuh"

NeuralNetwork::NeuralNetwork(vector<int> _topology)
{
	topology = _topology;
	topologySize = _topology.size();

	for (int i = 0; i < topology.size(); ++i)
	{
		Layer l = Layer(topology.at(i));
		layers.push_back(l);
	}

	for (int i = 0; i < topologySize - 1; ++i)
	{
		Matrix m = Matrix(topology.at(i), topology.at(i + 1), true);

		weightMatrices.push_back(m);
	}

	for (int i = 0; i < topology.at(topology.size() - 1); ++i)
	{
		errors.push_back(0.00);
	}
}

void NeuralNetwork::setCurrentInput(vector<double> _input)
{
	input = _input;

	for (int i = 0; i < input.size(); ++i)
	{
		layers.at(0).setVal(i, input.at(i));
	}

}

void NeuralNetwork::printToConsole()
{
	for (int i = 0; i < layers.size(); ++i)
	{
		cout << "LAYER: " << i << endl;

		if (i == 0)
		{
			Matrix m = layers.at(i).matrixifyVals();
			m.printToConsole();
		}
		else
		{
			Matrix m = layers.at(i).matrixifyActivatedVals();
			m.printToConsole();
		}
		cout << "--------------------------------------------------" << endl;
		if (i < layers.size() - 1)
		{
			cout << "Weight Matrix at index " << i << endl;
			getWeightMatrix(i).printToConsole();
		}
	}
}

void NeuralNetwork::feedForward()
{
	for (int i = 0; i < (layers.size() - 1); ++i)
	{
		Matrix a = getNeuronMatrix(i);

		if (i != 0)
		{
			a = getActivatedNeuronMatrix(i);
		}

		Matrix b = getWeightMatrix(i);
		Matrix c = utils::MultiplyMatrix(a, b).execute();

		for (int c_index = 0; c_index < c.getNumCols(); c_index++)
		{
			setNeuronValue(i + 1, c_index, c.getValue(0, c_index));
		}
	}
}

void NeuralNetwork::setErrors()
{
	if (target.size() == 0)
	{
		cerr << "No target for this neural network" << endl;
		assert(false);
	}

	if (target.size() != layers.at(layers.size() - 1).getNeurons().size())
	{
		cerr << "Target size is not the same as output layer size: " << layers.at(layers.size() - 1).getNeurons().size() << endl;
		assert(false);
	}

	error = 0.00;
	int outputLayerIndex = layers.size() - 1;
	vector<Neuron> outputNeurons = layers.at(outputLayerIndex).getNeurons();
	for (int i = 0; i < target.size(); i++)
	{
		//double tempErr = (outputNeurons.at(i)->getActivatedVal() - target.at(i));
		double tempErr = (outputNeurons.at(i).getActivatedVal() - target.at(i));
		errors.at(i) = tempErr;
		error += pow(tempErr, 2); //added pow to this line
	}

	error = 0.5 * error; //newaddition

	historicalErrors.push_back(error);
}

void NeuralNetwork::backPropagation()
{
	vector<Matrix> newWeights;
	Matrix gradients;


	//output layer to hidden layer
	int outputLayerIndex = layers.size() - 1;
	Matrix derivedValuesYToZ = layers.at(outputLayerIndex).matrixifyDerivedVals();
	Matrix gradientsYToZ = Matrix(1, layers.at(outputLayerIndex).getNeurons().size(), false);
	for (int i = 0; i < errors.size(); ++i)
	{
		double d = derivedValuesYToZ.getValue(0, i);
		double e = errors.at(i);
		double g = d * e;
		gradientsYToZ.setValue(0, i, g);
	}

	int lastHiddenLayerIndex = outputLayerIndex - 1;
	Layer lastHiddenLayer = layers.at(lastHiddenLayerIndex);
	Matrix weightOutputToHidden = weightMatrices.at(outputLayerIndex - 1);
	Matrix deltaOutputToHidden = utils::MultiplyMatrix(gradientsYToZ.transpose(), lastHiddenLayer.matrixifyActivatedVals()).execute().transpose();
	Matrix newWeightsOutputToHidden = Matrix(deltaOutputToHidden.getNumRows(), deltaOutputToHidden.getNumCols(), false);

	for (int r = 0; r < deltaOutputToHidden.getNumRows(); r++)
	{
		for (int c = 0; c < deltaOutputToHidden.getNumCols(); c++)
		{
			double originalWeight = weightOutputToHidden.getValue(r, c);
			double deltaWeight = deltaOutputToHidden.getValue(r, c);
			newWeightsOutputToHidden.setValue(r, c, (originalWeight - deltaWeight));
		}
	}

	newWeights.push_back(newWeightsOutputToHidden);
	gradients = Matrix(gradientsYToZ.getNumRows(), gradientsYToZ.getNumCols(), false);

	for (int r = 0; r < gradientsYToZ.getNumRows(); r++)
	{
		for (int c = 0; c < gradientsYToZ.getNumCols(); c++)
		{
			gradients.setValue(r, c, gradientsYToZ.getValue(r, c));
		}
	}

	//last hidden to input layer
	for (int i = outputLayerIndex - 1; i > 0; --i)
	{
		Layer l = layers.at(i);
		Matrix derivedHidden = l.matrixifyDerivedVals();
		Matrix derivedGradients = Matrix(1, l.getNeurons().size(), false);
		Matrix weightMatrix = weightMatrices.at(i);
		Matrix activatedHidden = l.matrixifyActivatedVals();
		Matrix originalWeight = weightMatrices.at(i - 1);

		for (int r = 0; r < weightMatrix.getNumRows(); r++)
		{
			double sum = 0.00;
			for (int c = 0; c < weightMatrix.getNumCols(); c++)
			{
				double p = gradients.getValue(0, c) * weightMatrix.getValue(r, c);
				sum += p;
			}

			double g = sum * activatedHidden.getValue(0, r);
			derivedGradients.setValue(0, r, g);
		}

		Matrix leftNeurons = (i - 1) == 0 ? layers.at(0).matrixifyVals() : layers.at(i - 1).matrixifyActivatedVals();

		Matrix deltaWeights = utils::MultiplyMatrix(derivedGradients.transpose(), leftNeurons).execute().transpose();

		Matrix newWeightsHidden = Matrix(deltaWeights.getNumRows(), deltaWeights.getNumCols(), false);

		for (int r = 0; r < newWeightsHidden.getNumRows(); r++)
		{
			for (int c = 0; c < newWeightsHidden.getNumCols(); c++)
			{
				double w = originalWeight.getValue(r, c);
				double d = deltaWeights.getValue(r, c);
				double n = w - d;
				newWeightsHidden.setValue(r, c, n);
			}
		}

		gradients = Matrix(derivedGradients.getNumRows(), derivedGradients.getNumCols(), false);

		for (int r = 0; r < derivedGradients.getNumRows(); r++)
		{
			for (int c = 0; c < derivedGradients.getNumCols(); c++)
			{
				gradients.setValue(r, c, derivedGradients.getValue(r, c));
			}
		}

		newWeights.push_back(newWeightsHidden);
	}

	reverse(newWeights.begin(), newWeights.end());

	weightMatrices = newWeights;

}

void NeuralNetwork::printInputToConsole()
{
	for (int i = 0; i < this->input.size(); ++i)
	{
		cout << this->input.at(i) << "\t";
	}
	cout << endl;
}

void NeuralNetwork::printOutputToConsole()
{
	int indexOfOutputLayer = layers.size() - 1;
	Matrix outputValues = layers.at(indexOfOutputLayer).matrixifyActivatedVals();
	for (int c = 0; c < outputValues.getNumCols(); c++)
	{
		cout << outputValues.getValue(0, c) << "\t";
	}
	cout << endl;
}

void NeuralNetwork::printTargetToConsole()
{
	for (int i = 0; i < this->target.size(); ++i)
	{
		cout << this->target.at(i) << "\t";
	}
	cout << endl;
}

void NeuralNetwork::printHistoricalErrors()
{
	for (int i = 0; i < this->historicalErrors.size(); ++i)
	{
		cout << this->historicalErrors.at(i);
		if (i != this->historicalErrors.size() - 1)
		{
			cout << ",";
		}
	}
	cout << endl;
}