#include <iostream>
#include <chrono>
#include "Neuron.cuh"
#include "Matrix.cuh"
#include "NeuralNetwork.cuh"

using namespace std;
using namespace std::chrono;

int main(int argc, char** argv)
{
	//test neurons
	/*
	Neuron* n = new Neuron(0.9);
	cout << "Val: " << n->getVal() << endl;
	cout << "ActivatedVal: " << n->getActivatedVal() << endl;
	cout << "DerivedVal: " << n->getDerivedVal() << endl;
	*/

	//test random matrix weights creation and transposing
	/*
	Matrix* m = new Matrix(3, 2, true);
	m->printToConsole();

	cout << "--------------------------------------------------" << endl;

	Matrix* mT = m->transpose();
	mT->printToConsole();
	*/

	//test network creation with input
	/*
	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	vector<double> input;
	input.push_back(1.0);
	input.push_back(0.0);
	input.push_back(1.0);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);

	nn->printToConsole();
	*/

	//check feedforward is calculating correctly through each layer
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(1);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->feedForward();
	nn->printToConsole();
	*/

	//check error calculations
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->setCurrentTarget(input);
	nn->feedForward();
	nn->setErrors();

	nn->printToConsole();

	cout << "Total Error: " << nn->getTotalError() << endl;
	*/

	//test backprop
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->setCurrentTarget(input);

	//training process
	for (int i = 0; i < 100000; ++i)
	{
		cout << "Epoch: " << i + 1 << endl;
		nn->feedForward();
		nn->setErrors();
		cout << "Total Error: " << nn->getTotalError() << endl;
		nn->backPropagation();
	}
	*/
	auto start = high_resolution_clock::now();

	//Start Copy Area
	//---------------
	vector<double> input;
	input.push_back(0.204829);
	input.push_back(0.905264);
	input.push_back(0.908988);
	input.push_back(0.111249);
	input.push_back(0.204858);
	input.push_back(0.272447);
	input.push_back(0.413717);
	input.push_back(0.38753);
	input.push_back(0.36959);
	input.push_back(0.922271);
	input.push_back(0.789755);
	input.push_back(0.325534);
	input.push_back(0.634112);
	input.push_back(0.498457);
	input.push_back(0.21524);
	input.push_back(0.933576);
	input.push_back(0.921143);
	input.push_back(0.755182);
	input.push_back(0.116525);
	input.push_back(0.559233);
	input.push_back(0.57177);
	input.push_back(0.0726854);
	input.push_back(0.670817);
	input.push_back(0.119705);
	input.push_back(0.488141);
	input.push_back(0.361441);
	input.push_back(0.215676);
	input.push_back(0.721447);
	input.push_back(0.369415);
	input.push_back(0.0293743);
	input.push_back(0.147754);
	input.push_back(0.970896);
	input.push_back(0.250955);
	input.push_back(0.539908);
	input.push_back(0.605861);
	input.push_back(0.960779);
	input.push_back(0.635517);
	input.push_back(0.316769);
	input.push_back(0.960725);
	input.push_back(0.621318);
	input.push_back(0.965797);
	input.push_back(0.807346);
	input.push_back(0.570731);
	input.push_back(0.605002);
	input.push_back(0.554977);
	input.push_back(0.329614);
	input.push_back(0.684465);
	input.push_back(0.753524);
	input.push_back(0.863657);
	input.push_back(0.340875);
	input.push_back(0.431487);
	input.push_back(0.7814);
	input.push_back(0.98531);
	input.push_back(0.128381);
	input.push_back(0.0876298);
	input.push_back(0.139685);
	input.push_back(0.872101);
	input.push_back(0.998877);
	input.push_back(0.255069);
	input.push_back(0.724658);
	input.push_back(0.144958);
	input.push_back(0.31156);
	input.push_back(0.686715);
	input.push_back(0.0477682);
	input.push_back(0.765126);
	input.push_back(0.0244933);
	input.push_back(0.827679);
	input.push_back(0.072816);
	input.push_back(0.343121);
	input.push_back(0.424249);
	input.push_back(0.81941);
	input.push_back(0.412208);
	input.push_back(0.508354);
	input.push_back(0.138124);
	input.push_back(0.128154);
	input.push_back(0.673945);
	input.push_back(0.882555);
	input.push_back(0.508395);
	input.push_back(0.590559);
	input.push_back(0.0423979);
	input.push_back(0.320903);
	input.push_back(0.325571);
	input.push_back(0.151148);
	input.push_back(0.742018);
	input.push_back(0.90565);
	input.push_back(0.761725);
	input.push_back(0.621686);
	input.push_back(0.911977);
	input.push_back(0.898315);
	input.push_back(0.713556);
	input.push_back(0.0673958);
	input.push_back(0.44961);
	input.push_back(0.58746);
	input.push_back(0.824068);
	input.push_back(0.913278);
	input.push_back(0.842669);
	input.push_back(0.779844);
	input.push_back(0.0169593);
	input.push_back(0.453853);
	input.push_back(0.980303);

	vector<double> target;
	target.push_back(0.112477);
	target.push_back(0.707666);
	target.push_back(0.34463);
	target.push_back(0.350625);
	target.push_back(0.860009);
	target.push_back(0.612047);
	target.push_back(0.883199);
	target.push_back(0.222597);
	target.push_back(0.13941);
	target.push_back(0.175897);
	target.push_back(0.000857692);
	target.push_back(0.343907);
	target.push_back(0.627541);
	target.push_back(0.0594374);
	target.push_back(0.104905);
	target.push_back(0.0359212);
	target.push_back(0.905715);
	target.push_back(0.141927);
	target.push_back(0.996518);
	target.push_back(0.215232);
	target.push_back(0.0824562);
	target.push_back(0.745878);
	target.push_back(0.0143425);
	target.push_back(0.215253);
	target.push_back(0.320253);
	target.push_back(0.509083);
	target.push_back(0.0856378);
	target.push_back(0.201664);
	target.push_back(0.359278);
	target.push_back(0.628177);
	target.push_back(0.196674);
	target.push_back(0.123806);
	target.push_back(0.0389247);
	target.push_back(0.372505);
	target.push_back(0.727115);
	target.push_back(0.476662);
	target.push_back(0.511225);
	target.push_back(0.91862);
	target.push_back(0.652862);
	target.push_back(0.905527);
	target.push_back(0.365569);
	target.push_back(0.633777);
	target.push_back(0.840801);
	target.push_back(0.415222);
	target.push_back(0.520421);
	target.push_back(0.657444);
	target.push_back(0.421453);
	target.push_back(0.768645);
	target.push_back(0.186551);
	target.push_back(0.191828);
	target.push_back(0.503818);
	target.push_back(0.313275);
	target.push_back(0.294159);
	target.push_back(0.816052);
	target.push_back(0.804983);
	target.push_back(0.947408);
	target.push_back(0.229653);
	target.push_back(0.834781);
	target.push_back(0.958809);
	target.push_back(0.247835);
	target.push_back(0.700575);
	target.push_back(0.525473);
	target.push_back(0.132892);
	target.push_back(0.340281);
	target.push_back(0.550285);
	target.push_back(0.0206471);
	target.push_back(0.535477);
	target.push_back(0.601023);
	target.push_back(0.680661);
	target.push_back(0.735823);
	target.push_back(0.539517);
	target.push_back(0.0106388);
	target.push_back(0.890069);
	target.push_back(0.545499);
	target.push_back(0.393838);
	target.push_back(0.648125);
	target.push_back(0.036597);
	target.push_back(0.113843);
	target.push_back(0.388771);
	target.push_back(0.459056);
	target.push_back(0.875386);
	target.push_back(0.61228);
	target.push_back(0.734639);
	target.push_back(0.284857);
	target.push_back(0.440033);
	target.push_back(0.721332);
	target.push_back(0.90889);
	target.push_back(0.150754);
	target.push_back(0.0327202);
	target.push_back(0.677347);
	target.push_back(0.169406);
	target.push_back(0.065766);
	target.push_back(0.849176);
	target.push_back(0.557173);
	target.push_back(0.944951);
	target.push_back(0.617289);
	target.push_back(0.801286);
	target.push_back(0.895104);
	target.push_back(0.621062);
	target.push_back(0.0791005);

	vector<int> topology;
	topology.push_back(100);
	topology.push_back(94);
	topology.push_back(52);
	topology.push_back(53);
	topology.push_back(100);
	//---------------
	//aerA ypoC tratS

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->setCurrentTarget(target);

	//training process
	for (int i = 0; i < 10000; ++i)
	{
		cout << "Epoch: " << i + 1 << endl;
		nn->feedForward();
		nn->setErrors();
		cout << "Total Error: " << nn->getTotalError() << endl;
		nn->backPropagation();

		cout << "----------------------------------------" << endl;
		cout << "OUTPUT: ";
		nn->printOutputToConsole();

		cout << "TARGET: ";
		nn->printTargetToConsole();
		cout << "----------------------------------------" << endl;
		cout << endl;

	}

	//nn->printHistoricalErrors();

	delete nn;

	auto stop = high_resolution_clock::now();

	auto duration = duration_cast<microseconds>(stop - start);

	cout << "CUDA" << endl;
	cout << "Time taken by function: "
		<< duration.count() / 1000000 << " seconds" << endl;

	return 0;
}