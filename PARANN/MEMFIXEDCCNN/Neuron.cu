#include "hip/hip_runtime.h"
#include "Neuron.cuh"

//Constructor
Neuron::Neuron(double _val)
{
	val = _val;
	activate();
	derive();
}

void Neuron::activate()
{
	activatedVal = val / (1 + abs(val));
}

void Neuron::derive()
{
	derivedVal = activatedVal * (1 - activatedVal);
}

void Neuron::setVal(double _val)
{
	val = _val;
	activate();
	derive();
}