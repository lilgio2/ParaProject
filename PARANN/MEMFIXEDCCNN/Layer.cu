#include "Layer.cuh"

Layer::Layer(int size)
{
	this->size = size;

	for (int i = 0; i < size; ++i)
	{
		Neuron n = Neuron(0.00);
		neurons.push_back(n);
	}
}

Matrix Layer::matrixifyVals()
{
	Matrix m = Matrix(1, neurons.size(), false);
	for (int i = 0; i < neurons.size(); ++i)
	{
		m.setValue(0, i, neurons.at(i).getVal());
	}
	return m;
}

Matrix Layer::matrixifyActivatedVals()
{
	Matrix m = Matrix(1, neurons.size(), false);
	for (int i = 0; i < neurons.size(); ++i)
	{
		m.setValue(0, i, neurons.at(i).getActivatedVal());
	}
	return m;
}

Matrix Layer::matrixifyDerivedVals()
{
	Matrix m = Matrix(1, neurons.size(), false);
	for (int i = 0; i < neurons.size(); ++i)
	{
		m.setValue(0, i, neurons.at(i).getDerivedVal());
	}
	return m;
}